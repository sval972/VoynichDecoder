#include "hip/hip_runtime.h"
#include "StaticTranslator.h"
#include <atomic>
#include <sstream>

#ifdef __NVCC__
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Global static CUDA resources (thread-safe initialization)
namespace {
    std::atomic<bool> g_isInitialized{false};
    hipblasHandle_t g_cublasHandle = nullptr;
    
    // CUDA kernel for binary matrix multiplication
    __global__ void binaryMatrixMultiplyKernel(
        const int* inputMatrix,      // N x 27 matrix (flattened)
        const int* transformMatrix,  // 27 x 27 matrix (flattened) 
        int* resultMatrix,           // N x 27 result matrix (flattened)
        int numWords,                // N (number of words)
        int matrixSize               // 27 (matrix dimension)
    ) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        
        if (row < numWords && col < matrixSize) {
            int sum = 0;
            
            // Compute dot product for binary matrices
            for (int k = 0; k < matrixSize; ++k) {
                sum += inputMatrix[row * matrixSize + k] & transformMatrix[k * matrixSize + col];
            }
            
            // For binary matrices, any non-zero result becomes 1
            resultMatrix[row * matrixSize + col] = (sum > 0) ? 1 : 0;
        }
    }
    
    void initializeCudaResources() {
        if (g_isInitialized.load()) return;
        
        // Simple initialization - no mutex needed since we're not sharing resources
        if (!g_isInitialized.exchange(true)) {
            // Initialize cuBLAS (optional - not currently used)
            hipblasStatus_t status = hipblasCreate(&g_cublasHandle);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                g_isInitialized.store(false);
                throw std::runtime_error("Failed to create cuBLAS handle");
            }
        }
    }
    
    void cleanupCudaResources() {
        if (g_cublasHandle) {
            hipblasDestroy(g_cublasHandle);
            g_cublasHandle = nullptr;
        }
        g_isInitialized.store(false);
    }
}

void StaticTranslator::performMatrixMultiplicationCuda(
    const std::vector<std::vector<int>>& inputMatrix,
    const std::vector<std::vector<int>>& transformMatrix,
    std::vector<std::vector<int>>& resultMatrix
) {
    size_t numWords = inputMatrix.size();
    if (numWords == 0) return;
    
    if (numWords > 1000) {
        throw std::runtime_error("Word count exceeds maximum supported size for CUDA");
    }
    
    // Initialize CUDA if needed (lightweight - no shared memory allocation)
    initializeCudaResources();
    
    // Allocate per-operation device memory (no sharing between threads)
    int* d_inputMatrix = nullptr;
    int* d_transformMatrix = nullptr; 
    int* d_resultMatrix = nullptr;
    
    // Allocate device memory for this operation
    size_t inputSize = numWords * 27 * sizeof(int);
    size_t transformSize = 27 * 27 * sizeof(int);
    size_t resultSize = numWords * 27 * sizeof(int);
    
    hipError_t error;
    
    // Allocate input matrix
    error = hipMalloc(&d_inputMatrix, inputSize);
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA memory allocation failed (input): " + std::string(hipGetErrorString(error)));
    }
    
    // Allocate transform matrix  
    error = hipMalloc(&d_transformMatrix, transformSize);
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        throw std::runtime_error("CUDA memory allocation failed (transform): " + std::string(hipGetErrorString(error)));
    }
    
    // Allocate result matrix
    error = hipMalloc(&d_resultMatrix, resultSize);
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        hipFree(d_transformMatrix);
        throw std::runtime_error("CUDA memory allocation failed (result): " + std::string(hipGetErrorString(error)));
    }
    
    // Flatten matrices for CUDA
    std::vector<int> flatInput(numWords * 27);
    std::vector<int> flatTransform(27 * 27);
    std::vector<int> flatResult(numWords * 27);
    
    // Fill input matrix
    for (size_t i = 0; i < numWords; ++i) {
        for (size_t j = 0; j < 27; ++j) {
            flatInput[i * 27 + j] = inputMatrix[i][j];
        }
    }
    
    // Fill transform matrix
    for (size_t i = 0; i < 27; ++i) {
        for (size_t j = 0; j < 27; ++j) {
            flatTransform[i * 27 + j] = transformMatrix[i][j];
        }
    }
    
    // Copy data to device
    error = hipMemcpy(d_inputMatrix, flatInput.data(), inputSize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        hipFree(d_transformMatrix);
        hipFree(d_resultMatrix);
        throw std::runtime_error("CUDA memory copy failed (input): " + std::string(hipGetErrorString(error)));
    }
    
    error = hipMemcpy(d_transformMatrix, flatTransform.data(), transformSize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        hipFree(d_transformMatrix);
        hipFree(d_resultMatrix);
        throw std::runtime_error("CUDA memory copy failed (transform): " + std::string(hipGetErrorString(error)));
    }
    
    // Set up kernel launch parameters
    dim3 blockSize(16, 16);
    dim3 gridSize((27 + blockSize.x - 1) / blockSize.x, (numWords + blockSize.y - 1) / blockSize.y);
    
    // Launch kernel
    binaryMatrixMultiplyKernel<<<gridSize, blockSize>>>(
        d_inputMatrix, d_transformMatrix, d_resultMatrix, 
        static_cast<int>(numWords), 27
    );
    
    // Check for kernel launch errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        hipFree(d_transformMatrix);
        hipFree(d_resultMatrix);
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Wait for completion
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        hipFree(d_transformMatrix);
        hipFree(d_resultMatrix);
        throw std::runtime_error("CUDA kernel execution failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Copy result back to host
    error = hipMemcpy(flatResult.data(), d_resultMatrix, resultSize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        hipFree(d_inputMatrix);
        hipFree(d_transformMatrix);
        hipFree(d_resultMatrix);
        throw std::runtime_error("CUDA memory copy failed (result): " + std::string(hipGetErrorString(error)));
    }
    
    // Clean up device memory
    hipFree(d_inputMatrix);
    hipFree(d_transformMatrix);
    hipFree(d_resultMatrix);
    
    // Unflatten result matrix
    for (size_t i = 0; i < numWords; ++i) {
        for (size_t j = 0; j < 27; ++j) {
            resultMatrix[i][j] = flatResult[i * 27 + j];
        }
    }
}

// Helper functions for CUDA availability and device info
bool isCudaAvailable_impl() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}

std::string getCudaDeviceInfo_impl() {
    if (!isCudaAvailable_impl()) {
        return "CUDA not available";
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    std::ostringstream info;
    info << "Device: " << prop.name 
         << ", Compute Capability: " << prop.major << "." << prop.minor
         << ", Global Memory: " << (prop.totalGlobalMem / (1024 * 1024)) << " MB"
         << ", Multiprocessors: " << prop.multiProcessorCount;
    
    return info.str();
}

// Register cleanup function to be called at program exit
static struct CudaCleanup {
    ~CudaCleanup() {
        cleanupCudaResources();
    }
} g_cudaCleanup;

#else

void StaticTranslator::performMatrixMultiplicationCuda(
    const std::vector<std::vector<int>>& inputMatrix,
    const std::vector<std::vector<int>>& transformMatrix,
    std::vector<std::vector<int>>& resultMatrix
) {
    throw std::runtime_error("CUDA support was not compiled into this binary");
}

bool isCudaAvailable_impl() {
    return false;
}

std::string getCudaDeviceInfo_impl() {
    return "CUDA not compiled";
}

#endif