#include "hip/hip_runtime.h"
#include "StaticTranslator.h"
#include <atomic>
#include <sstream>

#ifdef __NVCC__
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Global static CUDA resources (thread-safe initialization)
namespace {
    std::atomic<bool> g_isInitialized{false};
    hipblasHandle_t g_cublasHandle = nullptr;
    
    // Pre-allocated memory pools for performance
    thread_local void* g_deviceInputPool = nullptr;
    thread_local void* g_deviceTransformPool = nullptr;
    thread_local void* g_deviceResultPool = nullptr;
    thread_local size_t g_poolSizeInput = 0;
    thread_local size_t g_poolSizeTransform = 0;
    thread_local size_t g_poolSizeResult = 0;
    
    // Maximum batch size for optimal GPU utilization
    constexpr size_t MAX_BATCH_WORDS = 10000;     // Process up to 10K words at once
    constexpr size_t MAX_BATCH_MAPPINGS = 50000; // Process up to 50K mappings at once (reasonable GPU memory usage)
    constexpr size_t MATRIX_DIM = 27;
    
    // Optimized CUDA kernel with shared memory and better memory access
    __global__ void binaryMatrixMultiplyKernel(
        const int* __restrict__ inputMatrix,      // N x 27 matrix (flattened)
        const int* __restrict__ transformMatrix,  // 27 x 27 matrix (flattened) 
        int* __restrict__ resultMatrix,           // N x 27 result matrix (flattened)
        int numWords,                             // N (number of words)
        int matrixSize                            // 27 (matrix dimension)
    ) {
        // Shared memory for transform matrix (27x27 = 729 ints = 2.9KB)
        __shared__ int sharedTransform[27 * 27];
        
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int tid = threadIdx.y * blockDim.x + threadIdx.x;
        int threadsPerBlock = blockDim.x * blockDim.y;
        
        // Collaboratively load transform matrix into shared memory
        for (int i = tid; i < 27 * 27; i += threadsPerBlock) {
            sharedTransform[i] = transformMatrix[i];
        }
        __syncthreads();
        
        if (row < numWords && col < matrixSize) {
            int sum = 0;
            
            // Compute dot product using shared memory transform matrix
            const int* inputRow = &inputMatrix[row * matrixSize];
            
            // Unroll loop for better performance (27 iterations)
            #pragma unroll
            for (int k = 0; k < 27; ++k) {
                sum += inputRow[k] & sharedTransform[k * 27 + col];
            }
            
            // For binary matrices, any non-zero result becomes 1
            resultMatrix[row * matrixSize + col] = (sum > 0) ? 1 : 0;
        }
    }
    
    // High-performance batch kernel for processing multiple mappings in parallel
    __global__ void batchMatrixMultiplyKernel(
        const int* __restrict__ inputMatrix,      // numWords x 27 matrix (same for all mappings)
        const int* __restrict__ transformBatch,   // numMappings x (27 x 27) matrices
        int* __restrict__ resultBatch,            // numMappings x (numWords x 27) results
        int numWords,                             // Number of words (typically 100)
        int numMappings                           // Number of mappings to process in batch
    ) {
        int mappingId = blockIdx.z;                              // Which mapping (0 to numMappings-1)
        int row = blockIdx.y * blockDim.y + threadIdx.y;         // Word index (0 to numWords-1)
        int col = blockIdx.x * blockDim.x + threadIdx.x;         // Output column (0 to 26)
        
        if (mappingId >= numMappings || row >= numWords || col >= 27) return;
        
        // Shared memory for this mapping's transform matrix (27x27 = 729 ints = 2.9KB)
        __shared__ int sharedTransform[27 * 27];
        
        int tid = threadIdx.y * blockDim.x + threadIdx.x;
        int threadsPerBlock = blockDim.x * blockDim.y;
        
        // Collaboratively load transform matrix for this mapping into shared memory
        const int* thisTransform = &transformBatch[mappingId * 27 * 27];
        for (int i = tid; i < 27 * 27; i += threadsPerBlock) {
            sharedTransform[i] = thisTransform[i];
        }
        __syncthreads();
        
        // Compute matrix multiplication for this specific mapping
        int sum = 0;
        const int* inputRow = &inputMatrix[row * 27];
        
        // Unroll the inner loop for better performance
        #pragma unroll
        for (int k = 0; k < 27; ++k) {
            sum += inputRow[k] & sharedTransform[k * 27 + col];
        }
        
        // Store result for this mapping
        int resultIndex = mappingId * (numWords * 27) + row * 27 + col;
        resultBatch[resultIndex] = (sum > 0) ? 1 : 0;
    }
    
    void initializeCudaResources() {
        if (g_isInitialized.load()) return;
        
        // Simple initialization - no mutex needed since we're not sharing resources
        if (!g_isInitialized.exchange(true)) {
            // Initialize cuBLAS (optional - not currently used)
            hipblasStatus_t status = hipblasCreate(&g_cublasHandle);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                g_isInitialized.store(false);
                throw std::runtime_error("Failed to create cuBLAS handle");
            }
        }
    }
    
    void ensureMemoryPools(size_t maxWords) {
        size_t inputSize = maxWords * MATRIX_DIM * sizeof(int);
        size_t transformSize = MATRIX_DIM * MATRIX_DIM * sizeof(int);
        size_t resultSize = maxWords * MATRIX_DIM * sizeof(int);
        
        // Allocate or reallocate input pool if needed
        if (g_poolSizeInput < inputSize) {
            if (g_deviceInputPool) hipFree(g_deviceInputPool);
            hipMalloc(&g_deviceInputPool, inputSize);
            g_poolSizeInput = inputSize;
        }
        
        // Allocate or reallocate transform pool if needed  
        if (g_poolSizeTransform < transformSize) {
            if (g_deviceTransformPool) hipFree(g_deviceTransformPool);
            hipMalloc(&g_deviceTransformPool, transformSize);
            g_poolSizeTransform = transformSize;
        }
        
        // Allocate or reallocate result pool if needed
        if (g_poolSizeResult < resultSize) {
            if (g_deviceResultPool) hipFree(g_deviceResultPool);
            hipMalloc(&g_deviceResultPool, resultSize);
            g_poolSizeResult = resultSize;
        }
    }
    
    void cleanupMemoryPools() {
        if (g_deviceInputPool) {
            hipFree(g_deviceInputPool);
            g_deviceInputPool = nullptr;
            g_poolSizeInput = 0;
        }
        if (g_deviceTransformPool) {
            hipFree(g_deviceTransformPool);
            g_deviceTransformPool = nullptr;
            g_poolSizeTransform = 0;
        }
        if (g_deviceResultPool) {
            hipFree(g_deviceResultPool);
            g_deviceResultPool = nullptr;
            g_poolSizeResult = 0;
        }
    }
    
    void cleanupCudaResources() {
        cleanupMemoryPools();
        if (g_cublasHandle) {
            hipblasDestroy(g_cublasHandle);
            g_cublasHandle = nullptr;
        }
        g_isInitialized.store(false);
    }
}

void StaticTranslator::performMatrixMultiplicationCuda(
    const std::vector<std::vector<int>>& inputMatrix,
    const std::vector<std::vector<int>>& transformMatrix,
    std::vector<std::vector<int>>& resultMatrix
) {
    size_t numWords = inputMatrix.size();
    if (numWords == 0) return;
    
    // Cap at maximum batch size for optimal GPU utilization
    if (numWords > MAX_BATCH_WORDS) {
        throw std::runtime_error("Word count exceeds maximum CUDA batch size");
    }
    
    // Initialize CUDA if needed
    initializeCudaResources();
    
    // Ensure memory pools are large enough
    ensureMemoryPools(numWords);
    
    // Calculate sizes
    size_t inputSize = numWords * MATRIX_DIM * sizeof(int);
    size_t transformSize = MATRIX_DIM * MATRIX_DIM * sizeof(int);
    size_t resultSize = numWords * MATRIX_DIM * sizeof(int);
    
    // Flatten matrices for CUDA (reuse vectors to avoid allocations)
    static thread_local std::vector<int> flatInput, flatTransform, flatResult;
    flatInput.resize(numWords * MATRIX_DIM);
    flatTransform.resize(MATRIX_DIM * MATRIX_DIM);
    flatResult.resize(numWords * MATRIX_DIM);
    
    // Fill input matrix (optimized)
    for (size_t i = 0; i < numWords; ++i) {
        const auto& row = inputMatrix[i];
        std::copy(row.begin(), row.end(), flatInput.begin() + i * MATRIX_DIM);
    }
    
    // Fill transform matrix (optimized)
    for (size_t i = 0; i < MATRIX_DIM; ++i) {
        const auto& row = transformMatrix[i];
        std::copy(row.begin(), row.end(), flatTransform.begin() + i * MATRIX_DIM);
    }
    
    // Use pre-allocated device memory pools
    int* d_inputMatrix = static_cast<int*>(g_deviceInputPool);
    int* d_transformMatrix = static_cast<int*>(g_deviceTransformPool);
    int* d_resultMatrix = static_cast<int*>(g_deviceResultPool);
    
    // Copy data to device (async would be better, but sync for simplicity)
    hipError_t error = hipMemcpy(d_inputMatrix, flatInput.data(), inputSize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA input copy failed: " + std::string(hipGetErrorString(error)));
    }
    
    error = hipMemcpy(d_transformMatrix, flatTransform.data(), transformSize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA transform copy failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Optimize kernel launch parameters for better occupancy
    dim3 blockSize(32, 8);  // 256 threads per block (better than 16x16=256)
    dim3 gridSize((MATRIX_DIM + blockSize.x - 1) / blockSize.x, 
                  (numWords + blockSize.y - 1) / blockSize.y);
    
    // Launch kernel
    binaryMatrixMultiplyKernel<<<gridSize, blockSize>>>(
        d_inputMatrix, d_transformMatrix, d_resultMatrix, 
        static_cast<int>(numWords), MATRIX_DIM
    );
    
    // Check for kernel launch errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Synchronize only when we need the result
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA kernel execution failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Copy result back to host
    error = hipMemcpy(flatResult.data(), d_resultMatrix, resultSize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA result copy failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Unflatten result matrix (optimized)
    for (size_t i = 0; i < numWords; ++i) {
        auto& row = resultMatrix[i];
        std::copy(flatResult.begin() + i * MATRIX_DIM, 
                  flatResult.begin() + (i + 1) * MATRIX_DIM, 
                  row.begin());
    }
}

// High-performance batch processing for multiple mappings
void StaticTranslator::performBatchMatrixMultiplicationCuda(
    const std::vector<std::vector<int>>& inputMatrix,
    const std::vector<std::vector<std::vector<int>>>& transformMatrices,
    std::vector<std::vector<std::vector<int>>>& resultMatrices
) {
    size_t numWords = inputMatrix.size();
    size_t numMappings = transformMatrices.size();
    
    if (numWords == 0 || numMappings == 0) return;
    
    // Cap at maximum batch sizes for optimal GPU utilization
    if (numWords > MAX_BATCH_WORDS) {
        throw std::runtime_error("Word count exceeds maximum CUDA batch size");
    }
    if (numMappings > MAX_BATCH_MAPPINGS) {
        throw std::runtime_error("Mapping count exceeds maximum CUDA batch size");
    }
    
    // Initialize CUDA if needed
    initializeCudaResources();
    
    // Calculate memory requirements
    size_t inputSize = numWords * MATRIX_DIM * sizeof(int);
    size_t transformBatchSize = numMappings * MATRIX_DIM * MATRIX_DIM * sizeof(int);
    size_t resultBatchSize = numMappings * numWords * MATRIX_DIM * sizeof(int);
    
    // Ensure memory pools are large enough for batch processing
    ensureMemoryPools(std::max(numWords, numMappings * numWords));
    
    // Allocate additional memory for transform and result batches if needed
    void* d_transformBatch = nullptr;
    void* d_resultBatch = nullptr;
    
    hipError_t error = hipMalloc(&d_transformBatch, transformBatchSize);
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA transform batch allocation failed: " + std::string(hipGetErrorString(error)));
    }
    
    error = hipMalloc(&d_resultBatch, resultBatchSize);
    if (error != hipSuccess) {
        hipFree(d_transformBatch);
        throw std::runtime_error("CUDA result batch allocation failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Prepare host data (reuse thread-local vectors for efficiency)
    static thread_local std::vector<int> flatInput, flatTransforms, flatResults;
    flatInput.resize(numWords * MATRIX_DIM);
    flatTransforms.resize(numMappings * MATRIX_DIM * MATRIX_DIM);
    flatResults.resize(numMappings * numWords * MATRIX_DIM);
    
    // Fill input matrix (same for all mappings)
    for (size_t i = 0; i < numWords; ++i) {
        const auto& row = inputMatrix[i];
        std::copy(row.begin(), row.end(), flatInput.begin() + i * MATRIX_DIM);
    }
    
    // Fill transform matrices batch
    for (size_t mapping = 0; mapping < numMappings; ++mapping) {
        size_t offset = mapping * MATRIX_DIM * MATRIX_DIM;
        for (size_t i = 0; i < MATRIX_DIM; ++i) {
            const auto& row = transformMatrices[mapping][i];
            std::copy(row.begin(), row.end(), flatTransforms.begin() + offset + i * MATRIX_DIM);
        }
    }
    
    // Copy data to device
    error = hipMemcpy(g_deviceInputPool, flatInput.data(), inputSize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        hipFree(d_transformBatch);
        hipFree(d_resultBatch);
        throw std::runtime_error("CUDA input copy failed: " + std::string(hipGetErrorString(error)));
    }
    
    error = hipMemcpy(d_transformBatch, flatTransforms.data(), transformBatchSize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        hipFree(d_transformBatch);
        hipFree(d_resultBatch);
        throw std::runtime_error("CUDA transform batch copy failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Configure 3D grid for batch processing
    dim3 blockSize(16, 8, 1);  // 128 threads per block (good for most GPUs)
    dim3 gridSize(
        (MATRIX_DIM + blockSize.x - 1) / blockSize.x,      // Columns
        (numWords + blockSize.y - 1) / blockSize.y,        // Rows  
        numMappings                                         // Mappings (Z dimension)
    );
    
    // Launch batch kernel - processes all mappings in parallel!
    batchMatrixMultiplyKernel<<<gridSize, blockSize>>>(
        static_cast<int*>(g_deviceInputPool),
        static_cast<int*>(d_transformBatch),
        static_cast<int*>(d_resultBatch),
        static_cast<int>(numWords),
        static_cast<int>(numMappings)
    );
    
    // Check for kernel launch errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(d_transformBatch);
        hipFree(d_resultBatch);
        throw std::runtime_error("CUDA batch kernel launch failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Synchronize to wait for completion
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(d_transformBatch);
        hipFree(d_resultBatch);
        throw std::runtime_error("CUDA batch kernel execution failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Copy results back to host
    error = hipMemcpy(flatResults.data(), d_resultBatch, resultBatchSize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        hipFree(d_transformBatch);
        hipFree(d_resultBatch);
        throw std::runtime_error("CUDA batch result copy failed: " + std::string(hipGetErrorString(error)));
    }
    
    // Clean up temporary device memory
    hipFree(d_transformBatch);
    hipFree(d_resultBatch);
    
    // Unflatten results for each mapping
    resultMatrices.resize(numMappings);
    for (size_t mapping = 0; mapping < numMappings; ++mapping) {
        resultMatrices[mapping].resize(numWords, std::vector<int>(MATRIX_DIM));
        size_t offset = mapping * numWords * MATRIX_DIM;
        
        for (size_t i = 0; i < numWords; ++i) {
            auto& row = resultMatrices[mapping][i];
            std::copy(flatResults.begin() + offset + i * MATRIX_DIM,
                      flatResults.begin() + offset + (i + 1) * MATRIX_DIM,
                      row.begin());
        }
    }
}

// Helper functions for CUDA availability and device info
bool isCudaAvailable_impl() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}

std::string getCudaDeviceInfo_impl() {
    if (!isCudaAvailable_impl()) {
        return "CUDA not available";
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    std::ostringstream info;
    info << "Device: " << prop.name 
         << ", Compute Capability: " << prop.major << "." << prop.minor
         << ", Global Memory: " << (prop.totalGlobalMem / (1024 * 1024)) << " MB"
         << ", Multiprocessors: " << prop.multiProcessorCount;
    
    return info.str();
}

// Register cleanup function to be called at program exit
static struct CudaCleanup {
    ~CudaCleanup() {
        cleanupCudaResources();
    }
} g_cudaCleanup;

#else

void StaticTranslator::performMatrixMultiplicationCuda(
    const std::vector<std::vector<int>>& inputMatrix,
    const std::vector<std::vector<int>>& transformMatrix,
    std::vector<std::vector<int>>& resultMatrix
) {
    throw std::runtime_error("CUDA support was not compiled into this binary");
}

bool isCudaAvailable_impl() {
    return false;
}

std::string getCudaDeviceInfo_impl() {
    return "CUDA not compiled";
}

#endif